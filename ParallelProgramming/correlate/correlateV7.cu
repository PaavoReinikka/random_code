#include "hip/hip_runtime.h"
#include "cp.h"
#include <math.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define TILE_WIDTH 16

inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)

inline int static divup(int a, int b) {
    return (a + b - 1)/b;
}

inline int static roundup(int a, int b) {
    return divup(a, b) * b;
}

/*

__global__ void mykernel(float *res, float *data, int ny, int nx) {

	int row1=(blockIdx.y*blockDim.y+threadIdx.y);
	int row2=(blockIdx.x*blockDim.x+threadIdx.x);
	if (row1>row2 || row2 >=ny) {return;}
	
	float cumsum = 0.0;
	for (int col=0;col<nx;++col) {
		cumsum+=data[row1*nx+col]*data[row2*nx+col];
	}
	res[row1*ny+row2]=cumsum;
}
*/

__global__ void mykernel(float* res, float *data, int nx)
{
	//everything is preprocessed to fit nicely into square blocks
	__shared__ float shared_1[TILE_WIDTH*TILE_WIDTH];
	__shared__ float shared_2[TILE_WIDTH*TILE_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int row1 = blockIdx.y * TILE_WIDTH + ty;//threadIdx.y;
	int row2 = blockIdx.x * TILE_WIDTH + tx;//threadIdx.x;
	
	//preprocessing quarantees otherwise we stay within bounds
	// and this takes the upper diagonal
	if(row1>row2+TILE_WIDTH) return;
	float cumsum = 0;

	for (int i = 0; i < nx/TILE_WIDTH; ++i) {
		shared_1[ty*TILE_WIDTH+tx] = data[row1*nx + i*TILE_WIDTH + tx];
		shared_2[ty*TILE_WIDTH+tx] = data[i*TILE_WIDTH + ty+row2*nx];
		__syncthreads();
		
		for (int j = 0; j < TILE_WIDTH; ++j)
			cumsum += shared_1[ty*TILE_WIDTH+j] * shared_2[j*TILE_WIDTH+tx];
			__syncthreads();
		}
		res[row1*nx+row2] = cumsum;
}
void correlate(int ny, int nx, const float* data, float* result) {

    int threadsY=TILE_WIDTH;
    int threadsX=TILE_WIDTH;
    float s,ss;
	int NX=roundup(nx,TILE_WIDTH);
	int NY=roundup(ny,TILE_WIDTH);
	
	if(NX>NY) NY=NX; else NX=NY;

	float *temp = new float[NX*NY];
	float *temp2 = new float[NY*NY];

    for (int row=0;row<ny;row++) {
		s=0.0;
		for (int col=0;col<nx;col++) {
			s=s+data[nx*row+col];
		}
		ss=0.0;
		s/=nx;
		for (int col=0;col<nx;col++) {
			double elem = data[nx*row+col]-s;
			ss=ss+pow(elem,2);
			temp[NX*row+col]=elem;
		}
		ss=1.0/sqrt(ss);
		for (int col=0;col<nx;col++) {
			temp[NX*row+col]=ss*temp[NX*row+col];
		}
		for (int col=nx;col<NX;col++) temp[NX*row+col]=0.0f;
    }

    for(int row=ny;row<NY;row++)
	for(int col=0;col<NX;col++) temp[NX*row+col]=0.0f;

    float *dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, NY * NX * sizeof(float)));

    float *rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, NY * NY * sizeof(float)));
    CHECK(hipMemcpy(dGPU, temp, NY * NX * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(threadsX, threadsY);
    dim3 dimGrid(divup(NY,threadsX), divup(NY, threadsY));

    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, NX);
    CHECK(hipGetLastError());

	//IF HAPPENED TO BE SQUARE MATRIX, AND DIVISIBLE BY TILE_WIDTH
	// DON'T BOTHER USING temp2 BUT COPY DIRECTLY TO result	
    if(nx==NY) {
	CHECK(hipMemcpy(result, rGPU, NY * NY * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipFree(dGPU));
	CHECK(hipFree(rGPU));
    } else {
	CHECK(hipMemcpy(temp2, rGPU, NY * NY * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipFree(dGPU));
	CHECK(hipFree(rGPU));
	for(int row=0;row<ny;row++) {
	    for(int col=row;col<ny;col++) {
		result[row*ny+col]=temp2[row*NY+col];
	    }
	}
    }

    delete [] temp;delete [] temp2;
}


