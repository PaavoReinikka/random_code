#include "hip/hip_runtime.h"
#include "cp.h"
#include <math.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>

inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "<< hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)

int static divup(int a, int b) {
    return (a + b - 1)/b;
}

__global__ void mykernel(float *res, float *data, int ny, int nx) {
	int row1=(blockIdx.y*blockDim.y+threadIdx.y);
	int row2=(blockIdx.x*blockDim.x+threadIdx.x);
	if (row1>row2 || row2>=ny || row1 >=ny) {return;}
	res[row1*ny+row2]=data[row1*nx]*data[row2*nx];

	for (int col=1;col<nx;col++) 
		res[row1*ny+row2]+=data[row1*nx+col]*data[row2*nx+col];
}

void correlate(int ny, int nx, const float* data, float* result) {

    int threadsY=16;
    int threadsX=16;
    float *temp = new float[ny*nx];
    
    for (int row=0;row<ny;row++) {
		float s=0.0;
		for (int col=0;col<nx;col++) {
			s=s+data[nx*row+col];
		}
		float ss=0.0;
		s/=nx;
		for (int col=0;col<nx;col++) {
			double elem = data[nx*row+col]-s;
			ss=ss+pow(elem,2);
			temp[nx*row+col]=elem;
		}
		ss=1.0/sqrt(ss);
		for (int col=0;col<nx;col++) {
			temp[nx*row+col]=ss*temp[nx*row+col];
		}
	}

	float *dGPU = NULL;
	CHECK(hipMalloc((void**)&dGPU, ny * nx * sizeof(float)));
        float *rGPU = NULL;
        CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float)));
        CHECK(hipMemcpy(dGPU, temp, ny * nx * sizeof(float), hipMemcpyHostToDevice));

	dim3 dimBlock(threadsY, threadsX);
	dim3 dimGrid(divup(ny,threadsX), divup(ny, threadsY));

	mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, ny, nx);
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipFree(dGPU));
        CHECK(hipFree(rGPU));
	
	delete [] temp;
}


