#include "hip/hip_runtime.h"
#include "is.h"
#include <math.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>

inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "<< hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)

int static divup(int a, int b) {
    return (a + b - 1)/b;
}

__global__ void window_kernel(const float *ss, int *utils, const float vz, const int ny, const int nx) {

	int w = threadIdx.x + blockIdx.x*blockDim.x+1;
	int h = threadIdx.y + blockIdx.y*blockDim.y+1;
	int size = w*h;
	int z = nx*ny;
	
	//if(w>nx || h>ny) return;
	
	float xi = 1.0/size;
	float yi = 1.0/(z - size);
	float xiyi = xi + yi;
	float yivzz = yi*vz*vz;
	float yivz2 = yi*vz*2;
	int NX = nx + 1;
	float loc_max = 0.0f;

	for (int y0=0;y0<ny-h+1;y0++) {
		int y1_NX=NX*(y0+h);
        int y0_NX=y0*NX;

		#pragma unroll
		for (int x0=0;x0<nx-w+1;x0++) {
			float vx = ss[y1_NX+x0+w]+ss[y0_NX+x0] - ss[y1_NX+x0] - ss[y0_NX+x0+w];
			float vxx = (xiyi*vx-yivz2);//vx*(xiyi*vx-yivz2)+yivzz;
			vxx = vx*vxx + yivzz;
            loc_max = loc_max>vxx?loc_max:vxx;
        }
    }

	atomicMax(&utils[size], __float_as_int(loc_max));
}


Result segment(int ny, int nx, const float* data) {
    // FIXME
    int threadsY=8;
    int threadsX=16;

    //basic blocks ("corners")
    float *ss= new float[(nx+1)*(ny+1)];
    for (int i=0;i<=nx;i++) ss[i]=0.0f;//Empty first row
    const int NX=nx+1;

    for(int y1=1;y1<ny+1;y1++) {
        ss[y1*NX]=0.0;//empty first elem
	int i = 3*((y1-1)*nx-1);

        for(int x1=1;x1<nx+1;x1++) {
	    float tmp = data[3*x1+i];
            //float tmp = vd[(y1-1)*nx+x1-1];
            tmp -= ss[(y1-1)*NX+x1-1];
            tmp += ss[(y1-1)*NX+x1];
            tmp += ss[y1*NX+x1-1];
            ss[y1*NX+x1] = tmp;
        }
    }

    int all=nx*ny+nx+ny+1;
    int z=nx*ny;

    const float vz=ss[all-1];

    float *dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, all * sizeof(float)));
    CHECK(hipMemcpy(dGPU, ss, all * sizeof(float), hipMemcpyHostToDevice));

	//Just to be sure, initialize the utils array
    int *init = new int[z];
    for (int i=0;i<z;i++) init[i]=0;
    int *rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, z * sizeof(int)));
    CHECK(hipMemcpy(rGPU, init, z * sizeof(int), hipMemcpyHostToDevice));


    dim3 dimBlock(threadsX, threadsY);
    dim3 dimGrid(divup(nx,threadsX), divup(ny, threadsY));

    window_kernel<<<dimGrid, dimBlock>>>(dGPU, rGPU, vz, ny, nx);
    CHECK(hipGetLastError());

    float *loc_max = new float[z];
    CHECK(hipMemcpy(loc_max, rGPU, z * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));

	//Poll the global maximum, and corresponding window
    float glob_max=0.0f;
    int glob_size=0;
    for(int i=1;i<z;i++) {
		if(loc_max[i]>glob_max) {
		    glob_max=loc_max[i];
			glob_size=i;
		}
    }

    //Search the specific window of given size
    Result result;
    int size=glob_size;
    const float xi = 1.0/size;
    const float yi = 1.0/(z-size);
    const float xiyi = xi + yi;
    const float yivzz = yi*vz*vz;
    const float yivz2 = yi*vz*2;

    for (int w=nx;w>0;w--) {
	while(size%w) w--;
        int h=size/w;

        for (int y0=ny-h;y0>-1;y0--) {
            int y1_NX=NX*(y0+h);
            int y0_NX=y0*NX;

            for (int x0=nx-w;x0>-1;x0--) {
                float vx = ss[y1_NX+x0+w] + ss[y0_NX+x0];
				vx -= (ss[y1_NX+x0] + ss[y0_NX+x0+w]);
                float new_max = vx*(xiyi*vx-yivz2)+yivzz;

				//exit directly from here
                if(new_max>=glob_max) {

                    int X0=x0;
                    int X1=x0+w;
                    int Y0=y0;
                    int Y1=y0+h;

                    float inner = (ss[NX*Y1+X1] + ss[NX*Y0+X0] -
                                               ss[NX*Y1+X0] - ss[NX*Y0+X1]);
                    float outer = vz-inner;
                    int x=(Y1-Y0)*(X1-X0);
                    int y=z-x;
                    float dx= x==0 ? 0 :1.0/x;
                    float dy= y==0 ? 0 :1.0/y;
                    inner*=dx;
                    outer*=dy;

                    result = Result { Y0, X0, Y1, X1,{outer,outer,outer}, {inner,inner,inner} };
		    		delete [] loc_max;
		    		delete [] ss;
					delete [] init;
		    		return result;//ugly exit, saves little time

                }
            }
        }
    }

    delete [] loc_max;
    delete [] ss;
    delete [] init;
    return result;
}


